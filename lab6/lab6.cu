#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>

#define A 480

__global__
void map_m1(int n, float* m1) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= n) {
		return;
	}
	m1[i] = 1 / tanh(sqrt(m1[i]));
}

__global__
void map_m2(int n, float* m2, float* m2_copy) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= n) {
		return;
	}
	
	if (i != 0) {
        m2[i] += m2_copy[i];
    }

    m2[i] = fabs(tan(m2[i]));
}

__global__
void merge(int n, float* m1, float* m2) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= n) {
		return;
	}
	
	m2[i] = (m1[i] < m2[i]) ? m1[i] : m2[i];
}

__global__
void sort(float* m2, int start, int end) {
	int j = start;
	while (j < end - 1) {         
	    if (m2[j + 1] < m2[j]) {
	        int tmp = m2[j];
	        m2[j] = m2[j + 1];
	        m2[j + 1] = tmp;
	        j = start;
	    } else j++;
	}
}

int main(int argc, char *argv[]) {
    int N;
    float X = 0;

    struct timeval T1, T2;
    long delta_ms, delta_ms_Generate=0, delta_ms_Map=0, delta_ms_Merge=0, delta_ms_Sort=0, delta_ms_Reduce=0;
    long time_stamp_Generate=0, time_stamp_Map=0, time_stamp_Merge=0, time_stamp_Sort=0, time_stamp_Reduce=0;
    long cuda_time_Map=0, cuda_time_Merge=0, cuda_time_Sort=0, time_cut=0;
    float e = 0.00001;

    N = atoi(argv[1]); /* N равен первому параметру командной строки */
    
    gettimeofday(&T1, NULL);
    
    for(int it = 0; it < 50; it++) {
    	int j, tmp;

        unsigned int my_seed[1];
        my_seed[0] = it;
        
        /* 1. Generate */
        float *m1 = (float *) malloc(sizeof(float) * N);
        float *m2 = (float *) malloc(sizeof(float) * N / 2);
        float *m2_copy = (float *) malloc(sizeof(float) * N / 2 + 1);
        m2_copy[0] = 0;
        
        for (j = 0; j < N; j++) {
            m1[j] = rand_r(my_seed) % A + 1;
        }
        for (j = 0; j < N / 2; j++) {
            m2[j] = rand_r(my_seed) % (9 * A + 1) + A;
            m2_copy[j + 1] = m2[j];
        }
        
        gettimeofday(&T2, NULL);   /* запомнить текущее время T2 */
        time_stamp_Generate =  1000*(T2.tv_sec - T1.tv_sec) + (T2.tv_usec - T1.tv_usec)/1000;
        delta_ms_Generate += (time_stamp_Generate - time_stamp_Reduce);
        
        /* 2. Map */
        /* Решить поставленную задачу, заполнить массив с результатами*/
        float* cm1, *cm2, *cm2_copy;
        hipMalloc(&cm1, sizeof(float) * N);
        hipMalloc(&cm2, sizeof(float) * N / 2);
        hipMalloc(&cm2_copy, sizeof(float) * N / 2 + 1);

		hipMemcpy(cm1, m1, sizeof(float) * N, hipMemcpyHostToDevice);
		hipMemcpy(cm2, m2, sizeof(float) * N / 2, hipMemcpyHostToDevice);
		hipMemcpy(cm2_copy, m2_copy, sizeof(float) * N / 2 +1, hipMemcpyHostToDevice); 
		
		gettimeofday(&T2, NULL);   /* запомнить текущее время T2 */
        time_stamp_Map =  1000*(T2.tv_sec - T1.tv_sec) + (T2.tv_usec - T1.tv_usec)/1000;
		
		map_m1<<<(N+255)/256, 256>>>(N, cm1);  
		map_m2<<<(N+255)/256, 256>>>(N / 2, cm2, cm2_copy);  
		
		gettimeofday(&T2, NULL);   /* запомнить текущее время T2 */
		time_cut= (1000*(T2.tv_sec - T1.tv_sec) + (T2.tv_usec - T1.tv_usec)/1000);
        cuda_time_Map +=  time_cut - time_stamp_Map;
		
		gettimeofday(&T2, NULL);   /* запомнить текущее время T2 */
        time_stamp_Map =  1000*(T2.tv_sec - T1.tv_sec) + (T2.tv_usec - T1.tv_usec)/1000;
        delta_ms_Map += (time_stamp_Map - time_stamp_Generate);
		
        /* 3. Merge */
        
        gettimeofday(&T2, NULL);   /* запомнить текущее время T2 */
        time_stamp_Merge =  1000*(T2.tv_sec - T1.tv_sec) + (T2.tv_usec - T1.tv_usec)/1000;
        
		merge<<<(N+255)/256, 256>>>(N / 2, cm1, cm2);  
		
		gettimeofday(&T2, NULL);   /* запомнить текущее время T2 */
		time_cut= (1000*(T2.tv_sec - T1.tv_sec) + (T2.tv_usec - T1.tv_usec)/1000);
        cuda_time_Merge +=  time_cut - time_stamp_Merge;
		
		gettimeofday(&T2, NULL);   /* запомнить текущее время T2 */ 	
        time_stamp_Merge =  1000*(T2.tv_sec - T1.tv_sec) + (T2.tv_usec - T1.tv_usec)/1000;
        delta_ms_Merge += ((1000*(T2.tv_sec - T1.tv_sec) + (T2.tv_usec - T1.tv_usec)/1000) - time_stamp_Map);

		/* 4. Sort */
        /* Отсортировать массив с результатами указанным методом */
        hipStream_t s;
        int step = 10;
        for(j = 0; j < N/2; j += step) {
        	hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
        	//printf("sorting [%d; %d]\n", j, min(N/2, j+step));
        	
        	gettimeofday(&T2, NULL);   /* запомнить текущее время T2 */
       		time_stamp_Sort =  1000*(T2.tv_sec - T1.tv_sec) + (T2.tv_usec - T1.tv_usec)/1000;
        	
        	sort<<<1, 1, 0, s>>>(cm2, j, min(N/2, j+step));
        	
        	gettimeofday(&T2, NULL);   /* запомнить текущее время T2 */
        	time_cut= (1000*(T2.tv_sec - T1.tv_sec) + (T2.tv_usec - T1.tv_usec)/1000);
        	cuda_time_Sort += time_cut - time_stamp_Sort;
        	
        	hipStreamDestroy(s);
        }
        
        hipDeviceSynchronize();
		hipMemcpy(m2, cm2, sizeof(float) * N / 2, hipMemcpyDeviceToHost);
        
        j = 0;
        while (j < (N / 2) - 1) {
            if (m2[j + 1] < m2[j]) {
                tmp = m2[j];
                m2[j] = m2[j + 1];
                m2[j + 1] = tmp;
                j = 0;
            } else j++;
        }
        
        gettimeofday(&T2, NULL);   /* запомнить текущее время T2 */
        time_stamp_Sort =  1000*(T2.tv_sec - T1.tv_sec) + (T2.tv_usec - T1.tv_usec)/1000;
        delta_ms_Sort += (time_stamp_Sort - time_stamp_Merge);
        
        /* 5. Reduce */

        float min = 0;
        j = 0;
        tmp = 0;

        while (min == 0) {
            min = (fabs(m2[j]) < e) ? m2[j] : 0;
            j++;
        }
        
        for (j = 0; j < N / 2; j++) {
            int temp = 0;

            temp = (int) (m2[j] / min);

            if (temp % 2 == 0) {
                X = X + sin(m2[j]);
            }
        }
        
        gettimeofday(&T2, NULL);   /* запомнить текущее время T2 */
        time_stamp_Reduce =  1000*(T2.tv_sec - T1.tv_sec) + (T2.tv_usec - T1.tv_usec)/1000;
        delta_ms_Reduce += (time_stamp_Reduce - time_stamp_Sort);
        
        hipFree(cm1);
        hipFree(cm2);
        hipFree(cm2_copy);
        free(m1);
        free(m2);
        free(m2_copy);
    }
    gettimeofday(&T2, NULL);   /* запомнить текущее время T2 */
    delta_ms =  1000*(T2.tv_sec - T1.tv_sec) + (T2.tv_usec - T1.tv_usec)/1000;
    printf("\nN=%d. Milliseconds passed after Generate: %ld\n", N, delta_ms_Generate); /* T2 -T1 */
    printf("\nN=%d. Milliseconds passed after Map: %ld\n", N, delta_ms_Map); /* T2 -T1 */
    printf("\nN=%d. Milliseconds passed after Map on GPU: %ld\n", N, cuda_time_Map); /* T2 -T1 */
    printf("\nN=%d. Milliseconds passed after Merge: %ld\n", N, delta_ms_Merge); /* T2 -T1 */
    printf("\nN=%d. Milliseconds passed after Merge on GPU: %ld\n", N, cuda_time_Merge); /* T2 -T1 */
    printf("\nN=%d. Milliseconds passed after Sort: %ld\n", N, delta_ms_Sort); /* T2 -T1 */
    printf("\nN=%d. Milliseconds passed after Sort on GPU: %ld\n", N, cuda_time_Sort); /* T2 -T1 */
    printf("\nN=%d. Milliseconds passed after Reduce: %ld\n", N, delta_ms_Reduce); /* T2 -T1 */
    printf("\nN=%d. Milliseconds passed: %ld\n", N, delta_ms); /* T2 -T1 */
    printf("\nX: %f\n", X);
    return 0;
}
