#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>

#define A 480

__global__
void map_m1(int n, float* m1) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= n) {
		return;
	}
	m1[i] = 1 / tanh(sqrt(m1[i]));
}

__global__
void map_m2(int n, float* m2, float* m2_copy) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= n) {
		return;
	}
	
	if (i != 0) {
        m2[i] += m2_copy[i];
    }

    m2[i] = fabs(tan(m2[i]));
}

__global__
void merge(int n, float* m1, float* m2) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= n) {
		return;
	}
	
	m2[i] = (m1[i] < m2[i]) ? m1[i] : m2[i];
}

__global__
void sort(float* m2, int start, int end) {
	int j = start;
	while (j < end - 1) {         
	    if (m2[j + 1] < m2[j]) {
	        int tmp = m2[j];
	        m2[j] = m2[j + 1];
	        m2[j + 1] = tmp;
	        j = start;
	    } else j++;
	}
}

int main(int argc, char *argv[]) {
    int N;
    float X = 0;

    struct timeval T1, T2;
    double delta_ms;
    float e = 0.00001;

    N = atoi(argv[1]); /* N равен первому параметру командной строки */
    
    gettimeofday(&T1, NULL);
    
    for(int it = 0; it < 50; it++) {
    	int j, tmp;

        unsigned int my_seed[1];
        my_seed[0] = it;
        
        /* 1. Generate */
        float *m1 = (float *) malloc(sizeof(float) * N);
        float *m2 = (float *) malloc(sizeof(float) * N / 2);
        float *m2_copy = (float *) malloc(sizeof(float) * N / 2 + 1);
        m2_copy[0] = 0;
        
        for (j = 0; j < N; j++) {
            m1[j] = rand_r(my_seed) % A + 1;
        }
        for (j = 0; j < N / 2; j++) {
            m2[j] = rand_r(my_seed) % (9 * A + 1) + A;
            m2_copy[j + 1] = m2[j];
        }
        
        /* 2. Map */
        /* Решить поставленную задачу, заполнить массив с результатами*/
        float* cm1, *cm2, *cm2_copy;
        hipMalloc(&cm1, sizeof(float) * N);
        hipMalloc(&cm2, sizeof(float) * N / 2);
        hipMalloc(&cm2_copy, sizeof(float) * N / 2 + 1);

		hipMemcpy(cm1, m1, sizeof(float) * N, hipMemcpyHostToDevice);
		hipMemcpy(cm2, m2, sizeof(float) * N / 2, hipMemcpyHostToDevice);
		hipMemcpy(cm2_copy, m2_copy, sizeof(float) * N / 2 +1, hipMemcpyHostToDevice); 
		
		map_m1<<<(N+255)/256, 256>>>(N, cm1);  
		map_m2<<<(N+255)/256, 256>>>(N / 2, cm2, cm2_copy);  
		
        /* 3. Merge */
		merge<<<(N+255)/256, 256>>>(N / 2, cm1, cm2);  

		/* 4. Sort */
        /* Отсортировать массив с результатами указанным методом */
        hipStream_t s;
        int step = 10;
        for(j = 0; j < N/2; j += step) {
        	hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
        	//printf("sorting [%d; %d]\n", j, min(N/2, j+step));
        	sort<<<1, 1, 0, s>>>(cm2, j, min(N/2, j+step));
        	hipStreamDestroy(s);
        }
        
        hipDeviceSynchronize();
		hipMemcpy(m2, cm2, sizeof(float) * N / 2, hipMemcpyDeviceToHost);
        
        j = 0;
        while (j < (N / 2) - 1) {
            if (m2[j + 1] < m2[j]) {
                tmp = m2[j];
                m2[j] = m2[j + 1];
                m2[j + 1] = tmp;
                j = 0;
            } else j++;
        }
        
        j = 0;
        while (j < (N/2) - 1)
        {
            if (m2[j+1] < m2[j])
            {
                tmp = m2[j];
                m2[j] = m2[j+1];
                m2[j+1] = tmp;
                j = 0;
            }
            else j++;
        }
        
        /* 5. Reduce */

        float min = 0;
        j = 0;
        tmp = 0;

        while (min == 0) {
            min = (fabs(m2[j]) < e) ? m2[j] : 0;
            j++;
        }
        
        for (j = 0; j < N / 2; j++) {
            int temp = 0;

            temp = (int) (m2[j] / min);

            if (temp % 2 == 0) {
                X = X + sin(m2[j]);
            }
        }
        
        hipFree(cm1);
        hipFree(cm2);
        hipFree(cm2_copy);
        free(m1);
        free(m2);
        free(m2_copy);
    }
    gettimeofday(&T2, NULL);   /* запомнить текущее время T2 */
    delta_ms =  1000*(T2.tv_sec - T1.tv_sec) + (T2.tv_usec - T1.tv_usec)/1000;
    printf("\nN=%d. Milliseconds passed: %lf\n", N, delta_ms); /* T2 -T1 */
    printf("\nX: %f\n", X);
    return 0;
}
